#include "hip/hip_runtime.h"
#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

void Ped::Model::setupHeatmapSeq()
{
 
        size_t agentSize = agents.size();
        size_t AgentBytes =  sizeof(int) * agentSize;
        size_t HeatmapBytes = SIZE * SIZE * sizeof(int);
        size_t ScaledHeatmapBytes = SCALED_SIZE * SCALED_SIZE * sizeof(int);
	
	int *hm = (int*)calloc(SIZE*SIZE, sizeof(int));
	int *shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	
	//heatmap = (int**)malloc(SIZE*sizeof(int*));
	//scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	//blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	hipHostMalloc((void **)&heatmap, SIZE*sizeof(int*));
	hipHostMalloc((void **)&scaled_heatmap, SCALED_SIZE*sizeof(int*));
	hipHostMalloc((void **)&blurred_heatmap, SCALED_SIZE*sizeof(int*));
	    // allocering av minne i device variabler
	hipHostMalloc((void **)&h_desX, AgentBytes);
	hipHostMalloc((void **)&h_desY, AgentBytes);
	//hipHostMalloc((void **)&h_heatmap, HeatmapBytes);
	//hipHostMalloc((void **)&h_scaled_heatmap, ScaledHeatmapBytes);
	//hipHostMalloc((void **)&h_blurred_heatmap, ScaledHeatmapBytes);

	hipMalloc((void **)&d_desX, AgentBytes);
	hipMalloc((void **)&d_desY, AgentBytes);
	hipMalloc((void **)&d_heatmap, HeatmapBytes);
	hipMalloc((void **)&d_scaled_heatmap, ScaledHeatmapBytes);
	hipMalloc((void **)&d_blurred_heatmap, ScaledHeatmapBytes);
	hipMemcpyAsync((void *)d_heatmap, (void *)*heatmap, HeatmapBytes, hipMemcpyHostToDevice);

	
	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}
}

// Updates the heatmap according to the agent positions
__device__ void cuda_update(int *d_desX, int *d_desY, int *d_heatmap, int *d_scaled_heatmap, int *d_blurred_heatmap, size_t agentSize)
{
    int id = threadIdx.x;

	for (int x = 0; x < SIZE; x++)
	{
		d_heatmap[id*SIZE + x] = (int)round(d_heatmap[id*SIZE + x] * 0.80);
	}

    __syncthreads();

	// Count how many agents want to go to each location
    for (int i = id; i < agentSize; i = i + 1024) {
        int x = d_desX[i];
		int y = d_desY[i];

	if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
	  {

	  }
	else {
	  // intensify heat for better color results TODO: AtomicAdd
	  atomicAdd(&d_heatmap[y*SIZE + x],40);
	}
      }
    
    __syncthreads();
}

__device__ void scale_map(int *d_heatmap, int *d_scaled_heatmap) {
    // Scale the data for visual representation

  int row = blockIdx.y * blockDim.y + threadIdx.y; //   32x32
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
        d_heatmap[row*SIZE + col] = d_heatmap[row*SIZE + col] < 255 ? d_heatmap[row*SIZE + col] : 255;
	      int value = d_heatmap[row*SIZE + col];
	
	for (int cellY = 0; cellY < CELLSIZE; cellY++)
	  {
	    for (int cellX = 0; cellX < CELLSIZE; cellX++)
	      {
		d_scaled_heatmap[SCALED_SIZE*(row * CELLSIZE + cellY) + col * CELLSIZE + cellX] = value;
	      }
	  }	        
    __syncthreads();
}
__device__ void apply_gaussian(int *d_scaled_heatmap, int *d_blurred_heatmap)
{
  __shared__ int s_scaled_heatmap[32 * 32];
	// Weights for blur filter
  int row = blockIdx.y * blockDim.y + threadIdx.y; //(blockID * 32 + threadID) => [0,1,2,3,4,5,6,7..., 31]... [128, 129,...,n]
  int col = blockIdx.x * blockDim.x + threadIdx.x; //
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};
	////////////////////////////////////
	//INIT SHARED HEATMAP /////////////
	///////////////////////////////////
	s_scaled_heatmap[threadIdx.y * 32 + threadIdx.x] = d_scaled_heatmap[row * SCALED_SIZE + col];
	__syncthreads();
#define WEIGHTSUM 273
	// Apply gaussian blurfilter
	if (row >= 2 && row + 5 < SCALED_SIZE -2 && col >= 2 && col + 5 < SCALED_SIZE -2) {
	  if((threadIdx.y >= 2 && threadIdx.y + 5 < blockDim.y - 2 && threadIdx.x >= 2 && threadIdx.x + 5 < blockDim.x - 2)) { //Se till att vi inte hamnar utanför index	    
	    int sum = 0;
	    for (int k = -2; k < 3; k++)
	      {
		for (int l = -2; l < 3; l++)
		  {
		    sum +=  w[2 + k][2 + l] * s_scaled_heatmap[blockDim.x * (threadIdx.y + k) + (threadIdx.x + l)];
		  }
	      }
	    int value = sum / WEIGHTSUM;
	    d_blurred_heatmap[row * SCALED_SIZE + col] = 0x00FF0000 | value << 24;
	    
	  }
	  else
	    { //Fetch from d_scaled_heatmap - this means we hit outside designated tile
	    int sum = 0;
	    for (int k = -2; k < 3; k++)
	      {
		for (int l = -2; l < 3; l++)
		  {
		    sum +=  w[2 + k][2 + l] * d_scaled_heatmap[SCALED_SIZE * (row + k) + (col + l)];
		  }
	      }
	    int value = sum / WEIGHTSUM;
	    d_blurred_heatmap[row * SCALED_SIZE + col] = 0x00FF0000 | value << 24;
	  }
	  
	}
}


__global__ void kernelA(int *d_desX, int *d_desY, int *d_heatmap, int *d_scaled_heatmap, int *d_blurred_heatmap, size_t agentSize) {
  cuda_update(d_desX, d_desY, d_heatmap, d_scaled_heatmap, d_blurred_heatmap, agentSize);
}

__global__ void kernelB(int *d_heatmap, int *d_scaled_heatmap) {
  scale_map(d_heatmap, d_scaled_heatmap);
}

__global__ void kernelC(int *d_scaled_heatmap, int *d_blurred_heatmap) {
  apply_gaussian(d_scaled_heatmap, d_blurred_heatmap);
}


void Ped::Model::cuda_updateHeatmapSeq(){
    ///// SKAPA DATA ATT LADDA IN /////
    size_t agentSize = agents.size();	
    ///////////////////////////////////////
    ///////////////////////////////////////
    ///// INLADDNING AV DATA /////
    // storlekarna att allokera
    size_t AgentBytes =  sizeof(int) * agentSize;  
    size_t ScaledHeatmapBytes = SCALED_SIZE * SCALED_SIZE * sizeof(int);
  
    ///// INIT DATA SOM SKA LADDAS IN /////

    for (int i = 0; i < agentSize; i++){
      h_desX[i] = agents[i]->getDesiredX();
      h_desY[i] = agents[i]->getDesiredY();
    }
    // koppiering av värden från Host till Device
    hipMemcpyAsync((void *)d_desX, (void *)h_desX, AgentBytes, hipMemcpyHostToDevice);
    hipMemcpyAsync((void *)d_desY, (void *)h_desY, AgentBytes, hipMemcpyHostToDevice);
    
    //    hipMemcpyAsync((void *)d_scaled_heatmap, (void *)*scaled_heatmap, ScaledHeatmapBytes, hipMemcpyHostToDevice);
    //hipMemcpyAsync((void *)d_blurred_heatmap, (void *)*blurred_heatmap, ScaledHeatmapBytes, hipMemcpyHostToDevice);
    ///////////////////////////////////////

    ///// KALL AV KERNEL /////
    //id = 0, 5, 10, 15, 20
    dim3 dimBlock(32, 32); //32*32 threads per block = 1024
    dim3 dimGrid(SCALED_SIZE/dimBlock.y, SCALED_SIZE/dimBlock.x); //5120/128 = 40*40 = 1600 thread blocks
    dim3 dimGridB(SIZE/dimBlock.y, SIZE/dimBlock.x);
    kernelA<<<1, 1024>>>(d_desX, d_desY, d_heatmap, d_scaled_heatmap, d_blurred_heatmap, agentSize);

    kernelB<<<dimGridB, dimBlock>>>(d_heatmap, d_scaled_heatmap);
    
    kernelC<<<dimGrid, dimBlock>>>(d_scaled_heatmap, d_blurred_heatmap);

    // koppiering av värden från Device till Host.  
    //    hipMemcpyAsync((void *)*heatmap, (void *)d_heatmap, HeatmapBytes, hipMemcpyDeviceToHost);
    //hipMemcpyAsync((void *)*scaled_heatmap, (void *)d_scaled_heatmap, ScaledHeatmapBytes, hipMemcpyDeviceToHost);
    hipMemcpyAsync((void *)*blurred_heatmap, (void *)d_blurred_heatmap, ScaledHeatmapBytes, hipMemcpyDeviceToHost);
 
}
