#include "hip/hip_runtime.h"
#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// Updates the heatmap according to the agent positions
__device__ void cuda_update(int *d_desX, int *d_desY, int *d_heatmap, int *d_scaled_heatmap, int *d_blurred_heatmap, size_t agentSize)
{
    int id = threadIdx.x;

	for (int x = 0; x < SIZE; x++)
	{
		d_heatmap[id*SIZE + x] = (int)round(d_heatmap[id*SIZE + x] * 0.80);
	}

    __syncthreads();

	// Count how many agents want to go to each location
    if(id <= agentSize){
        int x = d_desX[id];
	int y = d_desY[id];

		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
		{

		}
		else {
		    // intensify heat for better color results TODO: AtomicAdd
		  atomicAdd(&d_heatmap[y*SIZE + x],40);
		}
    }

    __syncthreads();

    // Scale the data for visual representation
    for (int x = 0; x < SIZE; x++)
      {
        d_heatmap[id*SIZE + x] = d_heatmap[id*SIZE + x] < 255 ? d_heatmap[id*SIZE + x] : 255;
	      int value = d_heatmap[id*SIZE + x];
	
	for (int cellY = 0; cellY < CELLSIZE; cellY++)
	  {
	    for (int cellX = 0; cellX < CELLSIZE; cellX++)
	      {
		d_scaled_heatmap[SCALED_SIZE*(id * CELLSIZE + cellY) + x * CELLSIZE + cellX] = value;
	      }
	  }
      }
	        
    __syncthreads();
}
__device__ void apply_gaussian(int *d_scaled_heatmap, int *d_blurred_heatmap)
{
  __shared__ int s_scaled_heatmap[32 * 32];
  //__shared__ int s_blurred_heatmap[32 * 32];
	// Weights for blur filter
  int row = blockIdx.y * blockDim.y + threadIdx.y; //(blockID * 32 + threadID) => [0,1,2,3,4,5,6,7..., 127] [128, 129,...,n]
  int col = blockIdx.x * blockDim.x + threadIdx.x; //
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};
	////////////////////////////////////
	//INIT SHARED HEATMAP /////////////
	///////////////////////////////////
	s_scaled_heatmap[threadIdx.y * 32 + threadIdx.x] = d_scaled_heatmap[row * SCALED_SIZE + col];
	__syncthreads();
#define WEIGHTSUM 273
	// Apply gaussian blurfilter
	if (row >= 2 && row + 5 < SCALED_SIZE -2 && col >= 2 && col + 5 < SCALED_SIZE -2) {
	  if((threadIdx.y >= 2 && threadIdx.y + 5 < blockDim.y - 2 && threadIdx.x >= 2 && threadIdx.x + 5 < blockDim.x - 2)) { //Se till att vi inte hamnar utanför index	    
	    int sum = 0;
	    for (int k = -2; k < 3; k++)
	      {
		for (int l = -2; l < 3; l++)
		  {
		    sum +=  w[2 + k][2 + l] * s_scaled_heatmap[blockDim.x * (threadIdx.y + k) + (threadIdx.x + l)];
		  }
	      }
	    int value = sum / WEIGHTSUM;
	    d_blurred_heatmap[row * SCALED_SIZE + col] = 0x00FF0000 | value << 24;
	    
	  }
	  else
	    { //Fetch from d_scaled_heatmap - this means we hit outside designated tile
	    int sum = 0;
	    for (int k = -2; k < 3; k++)
	      {
		for (int l = -2; l < 3; l++)
		  {
		    sum +=  w[2 + k][2 + l] * d_scaled_heatmap[SCALED_SIZE * (row + k) + (col + l)];
		  }
	      }
	    int value = sum / WEIGHTSUM;
	    d_blurred_heatmap[row * SCALED_SIZE + col] = 0x00FF0000 | value << 24;	    
	  }
	  
	} 
	__syncthreads(); // Notera denna. Kanske inte behövlig.
}


__global__ void kernelA(int *d_desX, int *d_desY, int *d_heatmap, int *d_scaled_heatmap, int *d_blurred_heatmap, size_t agentSize) {
  cuda_update(d_desX, d_desY, d_heatmap, d_scaled_heatmap, d_blurred_heatmap, agentSize);
}

__global__ void kernelB(int *d_scaled_heatmap, int *d_blurred_heatmap) {
  apply_gaussian(d_scaled_heatmap, d_blurred_heatmap);
}


void Ped::Model::cuda_updateHeatmapSeq(){
    ///// SKAPA DATA ATT LADDA IN /////
    size_t agentSize = agents.size();
	int *h_desX;
	int *h_desY;
	int *d_desX;
	int *d_desY;

    int *h_heatmap;
    int *h_scaled_heatmap;
    int *h_blurred_heatmap;
    int *d_heatmap;
    int *d_scaled_heatmap;
    int *d_blurred_heatmap;
    ///////////////////////////////////////


    ///////////////////////////////////////

    ///// INLADDNING AV DATA /////

    // storlekarna att allokera
    size_t AgentBytes =  sizeof(int) * agentSize;
    size_t HeatmapBytes = SIZE * SIZE * sizeof(int);
    size_t ScaledHeatmapBytes = SCALED_SIZE * SCALED_SIZE * sizeof(int);

    // allocering av minne i device variabler
    hipHostMalloc((void **)&h_heatmap, HeatmapBytes);
    hipHostMalloc((void **)&h_scaled_heatmap, ScaledHeatmapBytes);
    hipHostMalloc((void **)&h_blurred_heatmap, ScaledHeatmapBytes);
    hipError_t errd = hipGetLastError();
    if ( errd != hipSuccess )
    {
       printf("CUDA Error in 'DEST!': %s\n", hipGetErrorString(errd));       
    }
    hipHostMalloc((void **)&h_desX, AgentBytes);
    hipHostMalloc((void **)&h_desY, AgentBytes);

  
    ///// INIT DATA SOM SKA LADDAS IN /////
    for (int i = 0; i < agentSize; i++){
      h_desX[i] = agents[i]->getDesiredX();
      h_desY[i] = agents[i]->getDesiredY();
    }
    for(int i = 0; i < SIZE; i++){
      for(int k = 0; k < SIZE; k++){
	
	h_heatmap[i*SIZE + k] = heatmap[i][k];

      }
    }


    hipError_t err1 = hipGetLastError();
    if ( err1 != hipSuccess )
    {
       printf("CUDA Error in 'allocering av minne i device variabler': %s\n", hipGetErrorString(err1));       
    }
    hipMalloc((void **)&d_desX, AgentBytes);
    hipMalloc((void **)&d_desY, AgentBytes);
    hipMalloc((void **)&d_heatmap, HeatmapBytes);
    hipMalloc((void **)&d_scaled_heatmap, ScaledHeatmapBytes);
    hipMalloc((void **)&d_blurred_heatmap, ScaledHeatmapBytes);

    // Record event för tidsgranskning
    float gpu_time;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // koppiering av värden från Host till Device
    hipMemcpy((void *)d_desX, (void *)h_desX, AgentBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_desY, (void *)h_desY, AgentBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_heatmap, (void *)h_heatmap, HeatmapBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_scaled_heatmap, (void *)h_scaled_heatmap, ScaledHeatmapBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_blurred_heatmap, (void *)h_blurred_heatmap, ScaledHeatmapBytes, hipMemcpyHostToDevice);
    hipError_t err2 = hipGetLastError();
    if ( err2 != hipSuccess )
    {
       printf("CUDA Error in 'koppiering av värden från Host till Device': %s\n", hipGetErrorString(err2));       
    }
    ///////////////////////////////////////

    ///// KALL AV KERNEL /////
    //id = 0, 5, 10, 15, 20
    dim3 dimBlock(32, 32); //32*32 threads per block = 1024
    dim3 dimGrid(SCALED_SIZE/dimBlock.y, SCALED_SIZE/dimBlock.x); //5120/128 = 40*40 = 1600 thread blocks  
    kernelA<<<1, 1024>>>(d_desX, d_desY, d_heatmap, d_scaled_heatmap, d_blurred_heatmap, agentSize);
    
    kernelB<<<dimGrid, dimBlock>>>(d_scaled_heatmap, d_blurred_heatmap);
    hipError_t err3 = hipGetLastError();
    if ( err3 != hipSuccess )
    {
       printf("CUDA Error in 'KALL AV KERNEL': %s\n", hipGetErrorString(err3));       
    }
    // koppiering av värden från Device till Host.  
    hipMemcpy((void *)h_heatmap, (void *)d_heatmap, HeatmapBytes, hipMemcpyDeviceToHost);
    hipMemcpy((void *)h_scaled_heatmap, (void *)d_scaled_heatmap, ScaledHeatmapBytes, hipMemcpyDeviceToHost);
    hipMemcpy((void *)h_blurred_heatmap, (void *)d_blurred_heatmap, ScaledHeatmapBytes, hipMemcpyDeviceToHost);

    hipError_t err4 = hipGetLastError();
    if ( err4 != hipSuccess )
    {
       printf("CUDA Error in 'koppiering av värden från Device till Host': %s\n", hipGetErrorString(err4));       
    }
    ///////////////////////////////////////

    ///// UPPDATERING AV VÄRDEN I MODEL /////
    
    /// OBS! detta är sjukt dummt och innefektivt.
    /// koppieringen bör gå dirrekt till model och detta ska inte behövas.
    /// Temp lösning för att kolla så allt funkar.
    for(int i = 0; i < SIZE; i++){
        for(int k = 0; k < SIZE; k++){
            heatmap[i][k] = h_heatmap[i*SIZE + k];
        }
    }

    for(int i = 0; i < SCALED_SIZE; i++){
        for(int k = 0; k < SCALED_SIZE; k++){    
            scaled_heatmap[i][k] = h_scaled_heatmap[i*SCALED_SIZE + k];
            blurred_heatmap[i][k] = h_blurred_heatmap[i*SCALED_SIZE + k];	    
	}
    }
    hipFree(d_desX);
    hipFree(d_desY);
    hipFree(d_heatmap);
    hipFree(d_scaled_heatmap);
    hipFree(d_blurred_heatmap);
    hipHostFree(h_desX);
    hipHostFree(h_desY);
    hipHostFree(heatmap);
    hipHostFree(scaled_heatmap);
    hipHostFree(blurred_heatmap);

    // PRINTA TID
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    std::cout << "\n gpu_time:  " << gpu_time;
    hipEventDestroy(start);
    hipEventDestroy(stop);






    /*
    delete [] h_desX;
    delete [] h_desY;
    delete [] h_heatmap;
    delete [] h_scaled_heatmap;
    delete [] h_blurred_heatmap;
    */
    
}