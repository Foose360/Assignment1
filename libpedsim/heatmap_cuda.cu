#include "hip/hip_runtime.h"
#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
#include "hip/hip_runtime.h"
#include ""

// Updates the heatmap according to the agent positions
__global__ void cuda_update(int *d_desX, int *d_desY, int **d_heatmap, int **d_scaled_heatmap, int **d_blurred_heatmap, size_t agentSize)
{
    int id = threadIdx.x;

	for (int x = 0; x < SIZE; x++)
	{
		d_heatmap[id][x] = (int)round(d_heatmap[id][x] * 0.80);
	}

    __syncthreads();

	// Count how many agents want to go to each location
    if(id <= agentSize){
        int x = d_desX[id];
		int y = d_desY[id];

		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
		{

		}
        else {
		    // intensify heat for better color results TODO: AtomicAdd
		    d_heatmap[y][x] += 40;
        }
    }

    __syncthreads();

	for (int x = 0; x < SIZE; x++)
	{
		d_heatmap[id][x] = d_heatmap[id][x] < 255 ? d_heatmap[id][x] : 255;
	}

    __syncthreads();

	// Scale the data for visual representation
		for (int x = 0; x < SIZE; x++){

		int value = d_heatmap[id][x];

		for (int cellY = 0; cellY < CELLSIZE; cellY++){
			for (int cellX = 0; cellX < CELLSIZE; cellX++){
				d_scaled_heatmap[id * CELLSIZE + cellY][x * CELLSIZE + cellX] = value;
			}
		}
	}

    __syncthreads();

	// Weights for blur filter
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};

#define WEIGHTSUM 273
	// Apply gaussian blurfilter		       
	for (int i = 2; i < CELLSIZE - 2; i++)
	{
		for (int j = 2; j < SCALED_SIZE - 2; j++)
		{
			int sum = 0;
			for (int k = -2; k < 3; k++)
			{
				for (int l = -2; l < 3; l++)
				{
					sum += w[2 + k][2 + l] * d_scaled_heatmap[i + id + k][j + l];
				}
			}
			int value = sum / WEIGHTSUM;
			d_blurred_heatmap[i][j] = 0x00FF0000 | value << 24;
		}
	}

    __syncthreads(); // Notera denna. Kanske inte behövlig.

}

void Ped::Model::cuda_updateHeatmapSeq(){
    ///// SKAPA DATA ATT LADDA IN /////
    size_t agentSize = agents.size();
	int h_desX[agentSize];
	int h_desY[agentSize];
	int *d_desX;
	int *d_desY;

    int h_heatmap[SIZE][SIZE];
    int h_scaled_heatmap[SCALED_SIZE][SCALED_SIZE];
    int h_blurred_heatmap[SCALED_SIZE][SCALED_SIZE];
    int **d_heatmap;
    int **d_scaled_heatmap;
    int **d_blurred_heatmap;

    ///////////////////////////////////////

    ///// INIT DATA SOM SKA LADDAS IN /////
	for (int i = 0; i < agentSize; i++){
		h_desX[i] = agents[i]->getDesiredX();
		h_desY[i] = agents[i]->getDesiredY();
	}

    for(int i = 0; i < SIZE; i++){
        for(int k = 0; k < SIZE; k++){

            h_heatmap[i][k] = heatmap[i][k];

        }
    }

    for(int i = 0; i < SCALED_SIZE; i++){       ///Detta borde inte vara nödvändigt, men safe:ar.
        for(int k = 0; k < SCALED_SIZE; k++){
    
            h_scaled_heatmap[i][k] = 0;
            h_blurred_heatmap[i][k] = 0;
        }
    }


    ///////////////////////////////////////

    ///// INLADDNING AV DATA /////

    // storlekarna att allokera
    size_t AgentBytes =  sizeof(int) * agentSize;
    size_t HeatmapBytes = SIZE * SIZE * sizeof(int);
    size_t ScaledHeatmapBytes = SCALED_SIZE * SCALED_SIZE * sizeof(int);

    // allocering av minne i device variabler
    hipHostMalloc((void **)&d_desX, AgentBytes);
    hipHostMalloc((void **)&d_desY, AgentBytes);

    hipError_t errd = hipGetLastError();
    if ( errd != hipSuccess )
    {
       printf("CUDA Error in 'DEST!': %s\n", hipGetErrorString(errd));       
    }

    hipHostMalloc((void **)&d_heatmap, HeatmapBytes);

    hipHostMalloc((void **)&d_scaled_heatmap, ScaledHeatmapBytes);
    hipHostMalloc((void **)&d_blurred_heatmap, ScaledHeatmapBytes);

    hipError_t err1 = hipGetLastError();
    if ( err1 != hipSuccess )
    {
       printf("CUDA Error in 'allocering av minne i device variabler': %s\n", hipGetErrorString(err1));       
    }


    // koppiering av värden från Host till Device
	hipMemcpy((void *)d_desX, (void *)h_desX, AgentBytes, hipMemcpyHostToDevice);
	hipMemcpy((void *)d_desY, (void *)h_desY, AgentBytes, hipMemcpyHostToDevice);

    hipMemcpy((void *)d_heatmap, (void *)h_heatmap, HeatmapBytes, hipMemcpyHostToDevice);

    hipMemcpy((void *)d_scaled_heatmap, (void *)h_scaled_heatmap, ScaledHeatmapBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_blurred_heatmap, (void *)h_blurred_heatmap, ScaledHeatmapBytes, hipMemcpyHostToDevice);

    hipError_t err2 = hipGetLastError();
    if ( err2 != hipSuccess )
    {
       printf("CUDA Error in 'koppiering av värden från Host till Device': %s\n", hipGetErrorString(err2));       
    }

    ///////////////////////////////////////

    ///// KALL AV KERNEL /////
	cuda_update<<<1, 1024>>>(d_desX, d_desY, d_heatmap, d_scaled_heatmap, d_blurred_heatmap, agentSize);

    hipError_t err3 = hipGetLastError();
    if ( err3 != hipSuccess )
    {
       printf("CUDA Error in 'KALL AV KERNEL': %s\n", hipGetErrorString(err3));       
    }

    // koppiering av värden från Device till Host.  
    hipMemcpy((void *)h_heatmap, (void *)d_heatmap, HeatmapBytes, hipMemcpyDeviceToHost);
    hipMemcpy((void *)h_scaled_heatmap, (void *)d_scaled_heatmap, ScaledHeatmapBytes, hipMemcpyDeviceToHost);
    hipMemcpy((void *)h_blurred_heatmap, (void *)d_blurred_heatmap, ScaledHeatmapBytes, hipMemcpyDeviceToHost);

    hipError_t err4 = hipGetLastError();
    if ( err4 != hipSuccess )
    {
       printf("CUDA Error in 'koppiering av värden från Device till Host': %s\n", hipGetErrorString(err4));       
    }

    ///////////////////////////////////////

    ///// UPPDATERING AV VÄRDEN I MODEL /////
    
    /// OBS! detta är sjukt dummt och innefektivt.
    /// koppieringen bör gå dirrekt till model och detta ska inte behövas.
    /// Temp lösning för att kolla så allt funkar.
    for(int i = 0; i < SIZE; i++){
        for(int k = 0; k < SIZE; k++){
            heatmap[i][k] = h_heatmap[i][k];
        }
    }

    for(int i = 0; i < SCALED_SIZE; i++){
        for(int k = 0; k < SCALED_SIZE; k++){
    
            scaled_heatmap[i][k] = h_scaled_heatmap[i][k];
            blurred_heatmap[i][k] = h_blurred_heatmap[i][k];
        }
    }
}